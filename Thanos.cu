
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <cmath>
#include <vector>
#include <stdlib.h>
#include <cstdint>
#include <random>
#include <algorithm>
#include <string>
#include <sstream>
#include <iterator>
#include <map>
#include <string.h>
#include <ctime>
#include <chrono>
#include <math.h>
#include <assert.h>
#include <cstring>

using namespace std::chrono;
using namespace std;

/* change the number if you want to partition with different size*/
#define PARTITION 4


/* !Kernel code that runs the cross-decomposition algorithm
*/
__global__ void CrossDecomposition_kernel(  uint8_t* orig_P, uint8_t* new_P, 
                                            uint64_t* cardi, uint64_t* new_cardi,
                                            const uint64_t num_nodes, const float h, uint64_t capacity,
                                            uint32_t* coo_row, uint32_t* coo_col, 
                                            uint64_t* row_ptr, bool is_divisible)
{
    uint64_t idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx < num_nodes){		
        uint64_t cur_node = idx;
        uint64_t connected_and_in_curpart[PARTITION] = {0};
        uint64_t degree = (row_ptr[cur_node+1] - row_ptr[cur_node]);

        for(int j=0; j<degree; j++){
            uint64_t cur_col_ptr = row_ptr[cur_node] + j;
            uint64_t cur_col = coo_col[cur_col_ptr];
            for (int cur_part = 0; cur_part < PARTITION; cur_part++) {
                if (orig_P[cur_col] == cur_part)
                    connected_and_in_curpart[cur_part] += 1;
            }
        }
        
        float cost[PARTITION] = {0};
        for (int i = 0; i < PARTITION; i++){
            cost[i] = h* connected_and_in_curpart[i] + 
                        (1-h)*(num_nodes - (cardi[i] + degree - connected_and_in_curpart[i]));
        } 

        //initialize arg_sort array
        uint8_t arg_sort[PARTITION];
        for (uint8_t i = 0; i < PARTITION; i++)
            arg_sort[i] = i;

        for (int i = 0; i < PARTITION-1; i++){       
            for (int j = 0; j < PARTITION-i-1; j++){
                if (cost[j] < cost[j+1]){
                    float temp = cost[j];
                    cost[j] = cost[j+1];
                    cost[j+1] = temp;
                    int temp2 = arg_sort[j];
                    arg_sort[j] = arg_sort[j+1];
                    arg_sort[j+1] = temp2;
                }
            }
        }

        unsigned long long int old_size;
        for (int i = 0; i < PARTITION; i++){
            if(!is_divisible){
                if(arg_sort[i]==PARTITION-1){
                    old_size = atomicAdd((unsigned long long int*)&new_cardi[arg_sort[i]],(unsigned long long int) 1);
                    if(old_size >= capacity+(num_nodes%PARTITION)){
                        old_size = atomicSub((unsigned int*)&new_cardi[arg_sort[i]],(unsigned int) 1);
                    }
                    else{
                        new_P[cur_node] = arg_sort[i];
                        break;
                    }
                }
                else{
                    old_size = atomicAdd((unsigned long long int*)&new_cardi[arg_sort[i]],(unsigned long long int) 1);
                    if(old_size >= capacity){
                        old_size = atomicSub((unsigned int*)&new_cardi[arg_sort[i]],(unsigned int) 1);
                    }
                    else{
                        new_P[cur_node] = arg_sort[i];
                        break;
                    }
                }
            }
            else{
                old_size = atomicAdd((unsigned long long int*)&new_cardi[arg_sort[i]],(unsigned long long int) 1);
                if(old_size >= capacity){
                    old_size = atomicSub((unsigned int*)&new_cardi[arg_sort[i]],(unsigned int) 1);
                }
                else{
                    new_P[cur_node] = arg_sort[i];
                    break;
                }
            }
        }
    }
}

/* !Kernel to count the number of edges to evaluate the quality of partition
*/
__global__ void evalEdges(uint8_t* P, 
                          uint32_t* coo_row, uint32_t* coo_col, 
                          uint64_t* edges_per_part, uint64_t coo_size)
{
    uint64_t idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<coo_size){
        uint32_t src = coo_row[idx];
        uint32_t dest = coo_col[idx];
        if(src != dest){
            uint8_t src_part = P[src];
            uint8_t dest_part = P[dest];
            atomicAdd( (unsigned long long int*)&edges_per_part[src_part*PARTITION+dest_part], (unsigned long long int) 1);
        }
    }
    return;
} 

/* !Simple function to check CUDA runtime error
*/
void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n",
            hipGetErrorString(result));
        assert(result == hipSuccess);
    }
}

/* !Thanos class definition
*/
class Thanos{
    private:
        float h = 0.9;
        bool is_divisible = true; 
        uint64_t num_nodes, edgecount, coo_size, capacity, rem;
        /*  following vectors are used to read files and create COO + CSR row pointer.
            They will be copied to GPU memories   */
        std::vector<uint32_t> edge_vec_src, edge_vec_dest; 
        std::vector<uint64_t> row_ptrs; 

        uint8_t *row_P, *col_P; //row and column partition arrays
        uint32_t* coo_row, *coo_col; //COO format
        uint64_t* row_ptr; //row pointer of CSR format
        uint64_t *row_cardi, *row_new_cardi, *col_cardi, *col_new_cardi; //cardinality arrays
        uint64_t* edges_per_part; //this var is for evaluating the partition quality

        hipError_t err;
        
        void readGraph_tsv(const char *filename);
        void readGraph_DARPA_CSR(   const char *filename,
                                    std::vector<uint32_t> &edge_vec_src,
                                    std::vector<uint32_t> &edge_vec_dest,
                                    std::vector<uint64_t> &row_ptrs,
                                    uint64_t &edgecount, uint64_t &nodecount);
        void AllocateGPUMem();
        void initMem();
        void initParts(uint8_t *P, uint64_t *cardi, const uint64_t num_nodes);
        void CrossDecomposition();
        void evaluatePartition();
        void printEdgesPerPar(uint64_t *edges_per_part);

    public:
        Thanos(){};
        Thanos(const char *filename);
        ~Thanos();

};

/* !Constructor for Thanos
    Construct Thanos with tsv file will run everyting for you
*/
Thanos::Thanos(const char *filename){
    readGraph_tsv(filename);
    AllocateGPUMem();
    initMem();
    evaluatePartition();
    CrossDecomposition();
    evaluatePartition();
}

/* !Destructor for Thanos. 
    Deallocates all the GPU memories
*/
Thanos::~Thanos(){
    hipFree(coo_row);
    hipFree(coo_col);
    hipFree(row_ptr);
    hipFree(row_P);
    hipFree(col_P);
    hipFree(row_cardi);
    hipFree(row_new_cardi);
    hipFree(col_cardi);
    hipFree(col_new_cardi);
    hipFree(edges_per_part);
}

/* !Host function to call the cross-decomposition kernel.
    You can change the boundary of for loop to control the number of iterations
*/
void Thanos::CrossDecomposition(){
    dim3 dimGrid(ceil(((float)num_nodes)/1024),1,1);
    dim3 dimBlock(1024,1,1);

    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    for(int i=0; i<3; i++){
        CrossDecomposition_kernel<<<dimGrid, dimBlock>>>(   row_P, col_P, col_cardi, col_new_cardi,
                                                            num_nodes, h, capacity,
                                                            coo_row, coo_col, 
                                                            row_ptr, is_divisible);

        CrossDecomposition_kernel<<<dimGrid, dimBlock>>>(   col_P, row_P, row_cardi, row_new_cardi,
                                                            num_nodes, h, capacity,
                                                            coo_row, coo_col, 
                                                            row_ptr, is_divisible);

        checkCuda(hipMemcpy(row_cardi, row_new_cardi, PARTITION*sizeof(uint64_t), hipMemcpyHostToHost));
        std::fill(row_new_cardi, row_new_cardi + PARTITION, 0);
        checkCuda(hipMemcpy(col_cardi, col_new_cardi, PARTITION*sizeof(uint64_t), hipMemcpyHostToHost));
        std::fill(col_new_cardi, col_new_cardi + PARTITION, 0);
    }
    
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error1: %s\n", hipGetErrorString(err));
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>( t2 - t1 ).count();
    cout<<"RUN Time in Sec(only kernel): "<<duration*pow(10,-6)<<endl;  
}

/* !Host function to call evalEdges kernel. After kernel call,
    it calls printEdgesPerPar function to show the result on the terminal
*/
void Thanos::evaluatePartition(){
    std::fill(edges_per_part, edges_per_part + PARTITION*PARTITION, 0);
    dim3 dimGrid0(ceil(((float)coo_size)/1024),1,1);
    dim3 dimBlock0(1024,1,1); //1024
    evalEdges<<<dimGrid0, dimBlock0>>>(row_P,coo_row, coo_col,
        edges_per_part, coo_size);
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error1: %s\n", hipGetErrorString(err));
    checkCuda(hipDeviceSynchronize());
    printEdgesPerPar(edges_per_part);
}


/* !Function to allocate all the memories required for Thanos on GPU.
*/
void Thanos::AllocateGPUMem(){
    
    checkCuda(hipMallocManaged((void**)&coo_row, coo_size*sizeof(uint32_t)));
    checkCuda(hipMallocManaged((void**)&coo_col, coo_size*sizeof(uint32_t)));
    checkCuda(hipMallocManaged((void**)&row_ptr, row_ptrs.size()*sizeof(uint64_t)));

    checkCuda(hipMallocManaged((void**)&row_P, num_nodes*sizeof(uint8_t)));
    checkCuda(hipMallocManaged((void**)&col_P, num_nodes*sizeof(uint8_t)));

    checkCuda(hipMallocManaged((void**)&row_cardi, PARTITION*sizeof(uint64_t)));
    checkCuda(hipMallocManaged((void**)&row_new_cardi, PARTITION*sizeof(uint64_t)));
    checkCuda(hipMallocManaged((void**)&col_cardi, PARTITION*sizeof(uint64_t)));
    checkCuda(hipMallocManaged((void**)&col_new_cardi, PARTITION*sizeof(uint64_t)));

    checkCuda(hipMallocManaged((void**)&edges_per_part, PARTITION*PARTITION*sizeof(uint64_t)));
    
}

/* !Initialize the memories that are allocated in function Allocate GPU Mem
*/
void Thanos::initMem(){
    checkCuda(hipMemcpy(coo_row, &edge_vec_src[0], coo_size*sizeof(uint32_t), hipMemcpyHostToHost));
    checkCuda(hipMemcpy(coo_col, &edge_vec_dest[0], coo_size*sizeof(uint32_t), hipMemcpyHostToHost));
    checkCuda(hipMemcpy(row_ptr, &row_ptrs[0], row_ptrs.size()*sizeof(uint64_t), hipMemcpyHostToHost));
    std::fill(row_cardi, row_cardi + PARTITION, 0);
    std::fill(row_new_cardi, row_new_cardi + PARTITION, 0);
    std::fill(col_cardi, col_cardi + PARTITION, 0);
    std::fill(col_new_cardi, col_new_cardi + PARTITION, 0);
    initParts(row_P, row_cardi, num_nodes);
    initParts(col_P, col_cardi, num_nodes);
    
}

/* !Function to read the graph and updates
    variables capacity and remainder(rem)
*/
void Thanos::readGraph_tsv(const char *filename){
    readGraph_DARPA_CSR(filename, edge_vec_src, edge_vec_dest, row_ptrs, edgecount, num_nodes);
    coo_size = edge_vec_src.size();
    cout<<"Reading Graph Done, num_nodes = "<<num_nodes<<" COO size: "<<coo_size<<endl;

    capacity = floor(num_nodes/PARTITION);
    rem = num_nodes%PARTITION;
    if(rem!=0){
        cout<<"Size of Each Partition is: "<<capacity<<endl;
        cout<<"Size of Last Partition is: "<<capacity+rem<<endl;
        is_divisible = false;
    }
    else{
        cout<<"N is divided perfectly"<<endl;
        cout<<"Size of Each Partition is: "<<capacity<<endl;
    }
}


/* !Helper function to read the TSV file of graph. 
    The TSV file must be in format of
    destination src weight
    The file has to be sorted with src first and
    destination also has to be sorted. 
*/
void Thanos::readGraph_DARPA_CSR(const char *filename,
                         std::vector<uint32_t> &edge_vec_src,
                         std::vector<uint32_t> &edge_vec_dest,
                         std::vector<uint64_t> &row_ptrs,
                         uint64_t &edgecount, uint64_t &nodecount)
{
    int key, val, weight;
    std::ifstream ss(filename);
    std::vector<std::pair<int, long long int>> temp_row_ptrs_vec;
    edgecount = 0;
    nodecount = 0;
    int prevkey = -1;
    if (ss.is_open() && ss.good()){
        while (ss >> val){
            ss >> key;
            ss >> weight;
            nodecount = std::max<int>(nodecount, key);
            key--;
            val--;
            //if(key < val) {
            if (prevkey != key){
                prevkey = key;
                edge_vec_src.push_back(key);
                edge_vec_dest.push_back(key);
                temp_row_ptrs_vec.push_back(std::pair<uint32_t, uint64_t>(key, edgecount));
                edgecount++;
            }
            edge_vec_src.push_back(key);
            edge_vec_dest.push_back(val);
            edgecount++;
            //}
        }
        ss.close();
    }

    uint64_t *temp_row_ptrs = new uint64_t[nodecount + 1];
    std::fill(temp_row_ptrs, temp_row_ptrs + nodecount, -1);
    temp_row_ptrs[nodecount] = edgecount;

    std::vector<std::pair<int, long long int>>::iterator begin = temp_row_ptrs_vec.begin();
    std::vector<std::pair<int, long long int>>::iterator end = temp_row_ptrs_vec.end();

    for (std::vector<std::pair<int, long long int>>::iterator it = begin; it != end; ++it)
        temp_row_ptrs[it->first] = it->second;

    long long int cur_val = edgecount;
    for (int i = nodecount; i >= 0; i--){
        long long int val = temp_row_ptrs[i];
        if (val < 0)
            temp_row_ptrs[i] = cur_val;
        else
            cur_val = val;
    }
    row_ptrs.insert(row_ptrs.begin(), temp_row_ptrs, temp_row_ptrs + nodecount + 1);
    delete[] temp_row_ptrs;
}

/* !Initialize the partition with uniform distribution and
    update the cardinality array
*/
void Thanos::initParts( uint8_t *P, uint64_t *cardi,
                        const uint64_t num_nodes)
{
    random_device rd;
    mt19937 mt(rd());
    uniform_int_distribution<int> dist(0, PARTITION - 1);
    for (size_t i = 0; i < num_nodes; i++){
        uint64_t part = dist(mt);
        P[i] = part;
        cardi[part]++;
    }
}

/* !This function nicely outputs the result of evaluation to the terminal
    if you want to see the number of edges in each partition and between partitions,
    uncomment the couts
*/
void Thanos::printEdgesPerPar(uint64_t *edges_per_part)
{
    uint32_t total_internal_edges = 0, total_external_edges = 0;
    cout << "********************************************************" << endl;
    map<pair<uint8_t, uint8_t>, bool> track;
    for (uint8_t i = 0; i < PARTITION; i++){
        for (uint8_t j = 0; j < PARTITION; j++){
            if (i == j){
                // cout << "Internal Edges for Partition " << (int)i << " :" << (edges_per_part[i * PARTITION + j]) / 2 << endl;
                total_internal_edges += edges_per_part[i * PARTITION + j] / 2;
            }
            else if (track.find(make_pair(i, j)) == track.end()){
                // cout << "Between "
                //  << "PARTITION " << (int)i << " and " << (int)j << " Edges: " << edges_per_part[i * PARTITION + j] << endl;
                total_external_edges += (edges_per_part[i * PARTITION + j]);
                track[make_pair(i, j)] = true;
                track[make_pair(j, i)] = true;
            }
        }
    }
    // cout << "--------------------------------------------------------" << endl;
    cout << "Total Internal Edges: " << total_internal_edges << endl;
    cout << "Total External Edges: " << total_external_edges << endl;
    cout << "********************************************************" << endl;
    return;
}






int main(int argc, char **argv){
    /*simply construct Thanos with providing the path for the tsv file. 
    It will run everything for you*/
    Thanos th(argv[1]);
    return 0;
}

